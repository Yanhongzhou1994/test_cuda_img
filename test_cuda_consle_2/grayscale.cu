#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"
#include "algorithm"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "vector"
#include "vector_functions.hpp"

__global__ void rgba_to_greyscale(const uchar4* const rgbaImage,unsigned char* const greyImage,
	int numRows,int numCols)
{
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	int grid_width = gridDim.x*blockDim.x;
	int index = index_y*grid_width + index_x;//index��ʾͼ���ָ��index
	greyImage[index] = .299f*rgbaImage[index].x + .587f*rgbaImage[index].y + .114f*rgbaImage[index].z;
	
}

extern "C"
void your_rgba_to_greyscale( uchar4* const d_rgbaImage,
	unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
	const int thread = 16;
	const dim3 blockSize(thread,thread,1);
	const dim3 gridSize(ceil(numRows/(float)thread),ceil(numCols/(float)thread),1);
	rgba_to_greyscale<<<gridSize, blockSize >>> (d_rgbaImage,d_greyImage,numRows,numCols);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}