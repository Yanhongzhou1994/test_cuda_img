#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include""
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_texture_types.h"
#include<math.h>

#include "hip/hip_runtime.h"
//#include "cpu_anim.h" //����texture��ʱ�����������ͷ�ļ�
#define size 256

texture<float, hipTextureType2D, hipReadModeElementType> texRef;

__global__ void transformKernel(float* input, float* output, int width, int height, float theta)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float u = x / (float)width;
	float v = y / (float)height;
	// ����ת��
	u -= 0.5f;
	v -= 0.5f;
	float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
	float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;
	int col = tu*width;
	int row = tv*height;
	//output[y*width + x] = input[0];
	output[y*width + x] = tex2D(texRef, tu, tv);
}

extern "C" 
void testTexture()
{
	int width = 3840, height = 1920;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray*cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);
	float*h_data = (float*)malloc(width*height * sizeof(float));
	for (int i = 0; i<height; ++i)
	{
		for (int j = 0; j<width; ++j)
		{
			h_data[i*width + j] = i*width + j;
		}
	}
	hipMemcpyToArray(cuArray, 0, 0, h_data, width*height * sizeof(float), hipMemcpyHostToDevice);
	texRef.addressMode[0] = hipAddressModeWrap;
	texRef.addressMode[1] = hipAddressModeWrap;
	texRef.filterMode = hipFilterModeLinear;
	texRef.normalized = true;
	hipBindTextureToArray(texRef, cuArray, channelDesc);
	float*output;
	hipMalloc(&output, width*height * sizeof(float));
	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
	float angle = 30;

	float *input = NULL;
	hipMalloc(&input, width*height * sizeof(float));
	hipMemcpy(input, h_data, width*height * sizeof(float), hipMemcpyHostToDevice);
	transformKernel << <dimGrid, dimBlock >> >(input, output, width, height, angle);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, NULL);

	for (int i = 0; i < 1000; i++)
	{
		transformKernel << <dimGrid, dimBlock >> >(input, output, width, height, angle);
		hipGetLastError();
	}
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float costtime;
	hipEventElapsedTime(&costtime, start, stop);
	printf("kernel run time: %f ms\n", costtime);

	float*hostPtr = (float*)malloc(sizeof(float)*width*height);
	hipMemcpy(hostPtr, output, sizeof(float)*width*height, hipMemcpyDeviceToHost);
	/*for (int i = 0; i<height; ++i)
	{
	for (int j = 0; j<width; ++j)
	{
	printf("%f\n", hostPtr[i*width + j]);
	}
	printf("\n");
	}*/
	free(hostPtr);
	hipFreeArray(cuArray);
	hipFree(output);
	system("pause");
}